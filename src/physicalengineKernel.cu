#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "physicalengine.h"


__global__ void calculateForceKernelFast(Particle* p, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    const float mass = p[idx].mass;
    const float mass2 = mass * mass;
    glm::vec3 F{};
    for (int i = idx, j = 0; j < N; ++j) {
        if (i == j) continue;
        glm::vec3 location_difference = p[j].location - p[i].location;
        float distance = glm::length(location_difference) + 0.1f;
        // printf("%f\n", distance);
        F += location_difference / (distance * distance * distance);
    }
    p[idx].force = F * mass2;
}

__global__ void calculateForceKernel(Particle* p, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    for (int i = idx, j = 0; j < N; ++j) {
        if (i == j) continue;
        glm::vec3 location_difference = p[j].location - p[i].location;
        float distance = glm::length(location_difference) + 0.1f;
        p[idx].force += p[i].mass * p[j].mass * location_difference / (distance * distance * distance);
    }
}

__global__ void updatePositionKernel(Particle* p, int N, float delta_time) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;
    int i = idx;
    p[i].acceleration = p[i].force / p[i].mass;
    p[i].velocity += p[i].acceleration * delta_time;
    p[i].location += p[i].velocity * delta_time;
    p[i].force = glm::vec3(0, 0, 0);
}

Particle* cu_p = nullptr;

void PhysicalEngine::nextCUDA(vector<Particle> &p, GLfloat delta_time) {
    static Particle *cu_p = nullptr;
    int N = p.size();
    if (!cu_p)
        hipMalloc((void**)&cu_p, N * sizeof(Particle));
    int M = 32;
    hipMemcpy(cu_p, &p[0], N * sizeof(Particle), hipMemcpyHostToDevice);
    calculateForceKernel<<<N / M + 1, M>>>(cu_p, N);
    updatePositionKernel<<<N / M + 1, M>>>(cu_p, N, delta_time);
    hipMemcpy(&p[0], cu_p, N * sizeof(Particle), hipMemcpyDeviceToHost);
    // hipFree(cu_p);
}

void PhysicalEngine::nextCUDAFast(vector<Particle> &p, GLfloat delta_time) {
    static Particle *cu_p = nullptr;
    int N = p.size();
    if (!cu_p)
        hipMalloc((void**)&cu_p, N * sizeof(Particle));
    int M = 32;
    hipMemcpy(cu_p, &p[0], N * sizeof(Particle), hipMemcpyHostToDevice);
    calculateForceKernelFast<<<N / M + 1, M>>>(cu_p, N);
    updatePositionKernel<<<N / M + 1, M>>>(cu_p, N, delta_time);
    hipMemcpy(&p[0], cu_p, N * sizeof(Particle), hipMemcpyDeviceToHost);
    // hipFree(cu_p);
}
